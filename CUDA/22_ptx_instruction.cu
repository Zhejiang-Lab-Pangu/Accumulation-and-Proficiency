//
// Created by zjlab on 9/26/23.
//


#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;


// 内置函数  __powf
__global__ void intrinsic(float *ptr){
    *ptr = __powf(*ptr, 2.0f);
}


// 标准函数 powf
__global__ void standard(float *ptr){
    *ptr = powf(*ptr, 2.0);
}

