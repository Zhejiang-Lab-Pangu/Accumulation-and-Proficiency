//
// Created by root on 10/18/23.
//



#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;


__global__ void bcast(int arg){
    int laneId = threadIdx.x & 0x1f;        // 与16进制的数等价于求余
//    printf("%d\n", laneId);

    int value = 0;
    if(laneId == 0){
        value = arg;
    }

    printf("threadidx(%d) -- value(%d)\n", threadIdx.x, value);

    __syncthreads();

    value = __shfl_sync(0xffffffff, value, 0, 32);
    if(value != arg){
        printf("thread %d failed\n", threadIdx.x);
    }
    printf("threadidx(%d) -- value(%d)\n", threadIdx.x, value);

}


__global__ void scan4(){

    int laneId = threadIdx.x % 32;
    int value = 31 - laneId;

    printf("threadidx(%d) -- value(%d)\n", threadIdx.x, value);

    __syncthreads();

    value = __shfl_up_sync(0xfffffff, value, 2, 32);
    printf("threadidx(%d) -- value(%d)\n", threadIdx.x, value);

}

__global__ void reduce(){
    int laneId = threadIdx.x & 0x1f;
    int value = 31 -laneId;

    printf("threadidx(%d) -- value(%d)\n", threadIdx.x, value);
    __syncthreads();

    value = __shfl_xor_sync(0xffffffff, value, 1, 32);
    printf("threadidx(%d) -- value(%d)\n", threadIdx.x, value);
}

int main(){

//    bcast<<<2, 32>>>(1234);
//    scan4<<<1, 32>>>();
    reduce<<<1, 32>>>();
    hipDeviceReset();

    return 0;
}