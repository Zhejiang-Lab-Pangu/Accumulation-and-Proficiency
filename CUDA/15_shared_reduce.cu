#include "hip/hip_runtime.h"
//
// Created by zjlab on 9/15/23.
//
//
// Created by zjlab on 9/13/23.
//
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "util.h"
#define DIM 1024


__global__ void reduceGmem(int *g_idata, int *g_odata, unsigned int n){

    // 块内的线程id
    unsigned int tid = threadIdx.x;
    // 全局的线程id
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(idx > n) return ;

    // 当前线程块的首地址
    int *idata = g_idata + blockIdx.x * blockDim.x;


    if(blockDim.x >= 1024 && tid < 512){
        idata[tid] += idata[tid+512];
    }
    __syncthreads();

    if(blockDim.x >= 512 && tid < 256){
        idata[tid] += idata[tid+256];
    }
    __syncthreads();

    if(blockDim.x >= 256 && tid < 128){
        idata[tid] += idata[tid+128];
    }
    __syncthreads();

    if(blockDim.x >= 128 && tid < 64){
        idata[tid] += idata[tid+64];
    }
    __syncthreads();


    if(tid<32){
        volatile int *vmem = idata;
        vmem[tid] += vmem[tid+32];
        vmem[tid] += vmem[tid+16];
        vmem[tid] += vmem[tid+8];
        vmem[tid] += vmem[tid+4];
        vmem[tid] += vmem[tid+2];
        vmem[tid] += vmem[tid+1];
    }

    if(tid == 0){
        g_odata[blockIdx.x] = idata[0];
    }
}


__global__ void reduceSmem(int *g_idata, int *g_odata, unsigned int n){


    __shared__ int smem[DIM];

    unsigned int tid = threadIdx.x;

    if(tid > n) return ;

    // 当前线程块的首地址
    int *idata = g_idata + blockIdx.x * blockDim.x;


    smem[tid] = idata[tid];
    __syncthreads();


    if(blockDim.x >= 1024 && tid < 512){
        smem[tid] += smem[tid+512];
    }
    __syncthreads();

    if(blockDim.x >= 512 && tid < 256){
        smem[tid] += smem[tid+256];
    }
    __syncthreads();

    if(blockDim.x >= 256 && tid < 128){
        smem[tid] += smem[tid+128];
    }
    __syncthreads();

    if(blockDim.x >= 128 && tid < 64){
        smem[tid] += smem[tid+64];
    }
    __syncthreads();


    if(tid<32){
        volatile int *vmem = smem;
        vmem[tid] += vmem[tid+32];
        vmem[tid] += vmem[tid+16];
        vmem[tid] += vmem[tid+8];
        vmem[tid] += vmem[tid+4];
        vmem[tid] += vmem[tid+2];
        vmem[tid] += vmem[tid+1];
    }

    if(tid == 0){
        g_odata[blockIdx.x] = smem[0];
    }
}


__global__ void reduceUnroll4Smem(int *g_idata, int *g_odata, unsigned int n){


    __shared__ int smem[DIM];

    // 这两个是同一个线程的两个索引，一个块内索引， 一个全局索引
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockDim.x * blockIdx.x * 4 + threadIdx.x;

    if(tid > n) return ;

    int tempSum = 0;
    if(idx + 3*blockDim.x <= n){
        int a1 = g_idata[idx];
        int a2 = g_idata[idx + blockDim.x];
        int a3 = g_idata[idx + blockDim.x * 2];
        int a4 = g_idata[idx + blockDim.x * 3];

        tempSum = a1 + a2 + a3 + a4;
    }

    // 将4个块的中存储的值相加copy到共享内存中，这里 tid 与 上面的 idx 相对应
    smem[tid] = tempSum;
    __syncthreads();


    if(blockDim.x >= 1024 && tid < 512){
        smem[tid] += smem[tid+512];
    }
    __syncthreads();

    if(blockDim.x >= 512 && tid < 256){
        smem[tid] += smem[tid+256];
    }
    __syncthreads();

    if(blockDim.x >= 256 && tid < 128){
        smem[tid] += smem[tid+128];
    }
    __syncthreads();

    if(blockDim.x >= 128 && tid < 64){
        smem[tid] += smem[tid+64];
    }
    __syncthreads();


    if(tid<32){
        volatile int *vmem = smem;
        vmem[tid] += vmem[tid+32];
        vmem[tid] += vmem[tid+16];
        vmem[tid] += vmem[tid+8];
        vmem[tid] += vmem[tid+4];
        vmem[tid] += vmem[tid+2];
        vmem[tid] += vmem[tid+1];
    }

    if(tid == 0){
        g_odata[blockIdx.x] = smem[0];
    }
}



int main(int argc,char **argv)
{

    initDevice(0);


    int Elemts = 1 << 24;
    int blocksize = 1024;

    dim3 block(blocksize, 1);
    dim3 grid((Elemts-1)/blocksize+1, 1);

    int nBytes = Elemts * sizeof(int);
    int *h_g_idata = new int[Elemts];
    int *h_o_idata = new int[grid.x];
    int *tmp = new int[Elemts];

    initialData_int(h_g_idata, Elemts);
    memcpy(tmp, h_g_idata, nBytes);

    // CPU
    int cpu_sum = 0;
    cpu_sum = recursiveReduce(tmp, Elemts);
    printf("cpu sum:%d \n", cpu_sum);


    int *d_idata = nullptr;
    int *d_odata = nullptr;

    CHECK(hipMalloc((void **)&d_idata, nBytes));
    CHECK(hipMalloc((void**)&d_odata, grid.x * sizeof(int)));

    // Global Mem
    CHECK(hipMemcpy(d_idata, h_g_idata, nBytes, hipMemcpyHostToDevice));
    reduceGmem<<<grid, block>>>(d_idata, d_odata, Elemts);
    hipDeviceSynchronize();
    CHECK(hipMemcpy(h_o_idata, d_odata, grid.x* sizeof(int), hipMemcpyDeviceToHost));
    int global_sum =0;
    for(int i=0; i<grid.x; i++){
        global_sum += h_o_idata[i];
    }
    printf("global sum:%d\n", global_sum);


    // Shared Mem
    CHECK(hipMemcpy(d_idata, h_g_idata, nBytes, hipMemcpyHostToDevice));
    reduceSmem<<<grid, block>>>(d_idata, d_odata, Elemts);
    hipDeviceSynchronize();
    CHECK(hipMemcpy(h_o_idata, d_odata, grid.x* sizeof(int), hipMemcpyDeviceToHost));
    int shared_sum =0;
    for(int i=0; i<grid.x; i++){
        shared_sum += h_o_idata[i];
    }
    printf("shared sum:%d\n", shared_sum);


    // global mem unroll + Shared Mem
    dim3 grid_unroll4(grid.x/4);
    CHECK(hipMemcpy(d_idata, h_g_idata, nBytes, hipMemcpyHostToDevice));
    reduceUnroll4Smem<<<grid_unroll4, block>>>(d_idata, d_odata, Elemts);
    hipDeviceSynchronize();
    CHECK(hipMemcpy(h_o_idata, d_odata, grid.x* sizeof(int), hipMemcpyDeviceToHost));
    int unroll_shared_sum =0;
    for(int i=0; i<grid.x; i++){
        unroll_shared_sum += h_o_idata[i];
    }
    printf("unroll shared sum:%d\n", unroll_shared_sum);

    return 0;
}