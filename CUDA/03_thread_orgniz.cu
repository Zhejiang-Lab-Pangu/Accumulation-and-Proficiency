#include "hip/hip_runtime.h"
//
// Created by root on 8/04/23.
//
#include <iostream>
using namespace std;
#include <mma.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "util.h"


/*
 *
 * 线程管理： dim3
 *
 * i = threadIdx + blockDim * blockIdx;
 *
 * */

void sumMatrix2D_CPU(float * MatA,float * MatB,float * MatC,int nx,int ny)
{
    float * a=MatA;
    float * b=MatB;
    float * c=MatC;
    for(int j=0;j<ny;j++)
    {
        for(int i=0;i<nx;i++)
        {
            c[i]=a[i]+b[i];
        }
        c+=nx;
        b+=nx;
        a+=nx;
    }
}


__global__ void printThreadIndex(float *A, const int nx, const int ny){
    int ix= threadIdx.x + blockIdx.x * blockDim.x;
    int iy= threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy * nx + ix;
    printf("thread_id(%d,%d) block_id(%d,%d) coordinate(%d,%d)"
           "global index %2d ival %2d\n",threadIdx.x,threadIdx.y, blockIdx.x,blockIdx.y, ix, iy, idx, A[idx]);
}

// 二维矩阵加法
__global__ void sumMatrix(float * MatA,float * MatB,float * MatC,int nx,int ny){
    int ix= threadIdx.x + blockIdx.x * blockDim.x;
    int iy= threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy * nx + ix;

    if(ix<nx && iy<ny){
        MatC[idx] = MatA[idx] + MatB[idx];
    }
}

void demo0(){
    initDevice(0);
    int nx=8, ny=6;
    int nxy= nx * ny;
    int nBytes= nxy * sizeof(float);

    //Malloc
    float * A_host = (float*)malloc(nBytes);
    initialData(A_host,nxy);
    printMatrix(A_host, nx, ny);

    //hipMalloc

    float *A_dev = nullptr;
    CHECK(hipMalloc((void **)&A_dev, nBytes));

    hipMemcpy(A_dev,A_host,nBytes,hipMemcpyHostToDevice);

    dim3 block(4, 2);
    dim3 grid((nx-1)/block.x + 1, (ny-1)/block.y + 1);

    printThreadIndex<<<grid,block>>>(A_dev,nx,ny);

    CHECK(hipDeviceSynchronize());
    hipFree(A_dev);
    free(A_host);

    hipDeviceReset();
}


void demo1(){
    printf("strating...\n");
    initDevice(0);

    int nx=1<<12;
    int ny=1<<12;
    int nxy=nx*ny;
    int nBytes=nxy*sizeof(float);

    //Malloc
    float* A_host=(float*)malloc(nBytes);
    float* B_host=(float*)malloc(nBytes);
    float* C_host=(float*)malloc(nBytes);
    float* C_from_gpu=(float*)malloc(nBytes);
    initialData(A_host,nxy);
    initialData(B_host,nxy);

    //hipMalloc
    float *A_dev=NULL;
    float *B_dev=NULL;
    float *C_dev=NULL;
    CHECK(hipMalloc((void**)&A_dev,nBytes));
    CHECK(hipMalloc((void**)&B_dev,nBytes));
    CHECK(hipMalloc((void**)&C_dev,nBytes));

    CHECK(hipMemcpy(A_dev,A_host,nBytes,hipMemcpyHostToDevice));
    CHECK(hipMemcpy(B_dev,B_host,nBytes,hipMemcpyHostToDevice));

    int dimx=32;
    int dimy=32;

    // cpu compute
    hipMemcpy(C_from_gpu,C_dev,nBytes,hipMemcpyDeviceToHost);
    double iStart=cpuSecond();
    sumMatrix2D_CPU(A_host,B_host,C_host,nx,ny);
    double iElaps=cpuSecond()-iStart;
    printf("CPU Execution Time elapsed %f sec\n",iElaps);


    // 2d block and 2d grid
    dim3 block_0(dimx,dimy);
    dim3 grid_0((nx-1)/block_0.x+1,(ny-1)/block_0.y+1);
    iStart=cpuSecond();
    sumMatrix<<<grid_0,block_0>>>(A_dev,B_dev,C_dev,nx,ny);
    CHECK(hipDeviceSynchronize());
    iElaps=cpuSecond()-iStart;
    printf("GPU Execution configuration<<<(%d,%d),(%d,%d)>>> Time elapsed %f sec\n", grid_0.x, grid_0.y, block_0.x, block_0.y, iElaps);
    CHECK(hipMemcpy(C_from_gpu,C_dev,nBytes,hipMemcpyDeviceToHost));
    checkResult(C_host,C_from_gpu,nxy);


    // 1d block and 1d grid
    dimx=32;
    dim3 block_1(dimx);
    dim3 grid_1((nxy-1)/block_1.x+1);
    iStart=cpuSecond();
    sumMatrix<<<grid_1,block_1>>>(A_dev,B_dev,C_dev,nx*ny ,1);
    CHECK(hipDeviceSynchronize());
    iElaps=cpuSecond()-iStart;
    printf("GPU Execution configuration<<<(%d,%d),(%d,%d)>>> Time elapsed %f sec\n", grid_1.x,grid_1.y,block_1.x,block_1.y,iElaps);
    CHECK(hipMemcpy(C_from_gpu,C_dev,nBytes,hipMemcpyDeviceToHost));
    checkResult(C_host,C_from_gpu,nxy);


    // 1d block and 2d grid
    dimx=32;
    dim3 block_2(dimx);
    dim3 grid_2((nx-1)/block_2.x+1,ny);
    iStart=cpuSecond();
    sumMatrix<<<grid_2,block_2>>>(A_dev,B_dev,C_dev,nx,ny);
    CHECK(hipDeviceSynchronize());
    iElaps=cpuSecond()-iStart;
    printf("GPU Execution configuration<<<(%d,%d),(%d,%d)>>> Time elapsed %f sec\n", grid_2.x,grid_2.y,block_2.x,block_2.y,iElaps);
    CHECK(hipMemcpy(C_from_gpu,C_dev,nBytes,hipMemcpyDeviceToHost));
    checkResult(C_host,C_from_gpu,nxy);


    hipFree(A_dev);
    hipFree(B_dev);
    hipFree(C_dev);
    free(A_host);
    free(B_host);
    free(C_host);
    free(C_from_gpu);
    hipDeviceReset();
}

int main()
{
//    demo0();
    cout << endl;
    demo1();
    return 0;
}

