#include "hip/hip_runtime.h"
//
// Created by zjlab on 9/27/23.
//
#include <iostream>
using namespace std;
#include "hipsparse.h"
#include "util.h"
#include "hip/hip_runtime.h"


/*
 * cuSparse中的数据类型
 *
 * 1.hipsparseMatDescr_t 描述稀疏矩阵的形状特征
 *
 *   (1) hipsparseDiagType_t     用于指示对角线元素是不是都是1(如果是的话，那API在执行的时候就不会去读写对角线元素)
 *          HIPSPARSE_DIAG_TYPE_NON_UNIT
 *          HIPSPARSE_DIAG_TYPE_UNIT
 *
 *   (2) hipsparseFillMode_t     运算中计算矩阵上三角或下三角部分
 *          HIPSPARSE_FILL_MODE_LOWER
 *          HIPSPARSE_FILL_MODE_UPPER
 *
 *   (3) hipsparseIndexBase_t    指标从0开始还是1开始
 *          HIPSPARSE_INDEX_BASE_ZERO
 *          HIPSPARSE_INDEX_BASE_ONE
 *
 * */



__global__ void gpu_print(float *mat, int n){
    printf("hello gpu\n");
    for(int i =0; i<n; i++){
        printf("%f\n", mat[i]);
    }
}

void demo0(){
    initDevice(0);


    float a[] = {
            1.0f, 0, 2.0f, 0,
            0, 3.0f, 0, 0,
            4.0f, 5.0f, 0, 0,
            0, 0, 7.0f, 8.0f
    };

    float csr_a[] = {1.0f, 4.0f, 3.0f, 5.0f, 2.0f, 7.0f, 8.0f};

    float b[] = {1.0f, 2.0f, 3.0f, 4.0f,
                 5.0f, 6.0f, 7.0f, 8.0f,
                 9.0f, 10.0f, 11.0f, 12.0f,
                 13.0f, 14.0f, 15.0f, 16.0f
    };

    cout << "sparse matrix dense format: " << endl;
    printMatrix(a, 4, 4);

    // host
    float *lhs = a, *rhs =b;
    float *res = new float [16];
    float *host_csr_lhs = csr_a;


    //device
    float *d_lhs, *d_rhs, *d_res;
    hipMalloc((void **)&d_lhs, 16 * sizeof(float));
    hipMalloc(&d_rhs, 16*sizeof(float));
    hipMalloc(&d_res, 16* sizeof(float));


    hipMemcpy(d_lhs, lhs, 16 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_rhs, rhs, 16 * sizeof(float), hipMemcpyHostToDevice);



    // 创建cuSPARSE库句柄
    hipsparseHandle_t handle;
    hipsparseCreate(&handle);


    // hipsparseCreateMatDescr和cusparseSetMat*配置矩阵某一属性
    hipsparseMatDescr_t dsr;
    hipsparseCreateMatDescr(&dsr);
    hipsparseSetMatType(dsr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(dsr, HIPSPARSE_INDEX_BASE_ZERO);



    int * nnzPerRowColumn = new int[4];
    int num=0;
    int * nnzperrow = nullptr;
    hipMalloc(&nnzperrow, sizeof(int) *4);

    // hipsparseSnnz: 统计稠密矩阵中各列和各行非零元素的数目， 以及总数
    CHECK_SPARSE(hipsparseSnnz(handle, HIPSPARSE_DIRECTION_ROW, 4, 4, dsr, d_lhs, 4, nnzperrow, &num));


    hipMemcpy(nnzPerRowColumn, nnzperrow, 16, hipMemcpyDeviceToHost);

    cout << "nonzero elem nums pre row:" << endl;
    for(int i=0; i<4; i++){
        cout << nnzPerRowColumn[i] << "  ";
    }
    cout << endl;
    cout << "none zero nums: " << num << endl;

    // host
    float * csr_lhs = new float(num);
    int *csr_lhs_col_idx = new int (num);
    int *csr_lhs_row_idx = new int(5);

    // device
    float *d_csr_lhs;
    int *d_csr_lhs_col_idx, *d_csr_lhs_row_idx;
    hipMalloc(&d_csr_lhs, num*sizeof(float));
    hipMalloc(&d_csr_lhs_col_idx, num * sizeof(int));
    hipMalloc(&d_csr_lhs_row_idx, 5 * sizeof(int));

    // dense转csr
    hipsparseSdense2csr(handle, 4, 4, dsr, d_lhs, 4, nnzperrow, d_csr_lhs, d_csr_lhs_row_idx, d_csr_lhs_col_idx);

    hipMemcpy(csr_lhs, d_csr_lhs, num* sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(csr_lhs_col_idx, d_csr_lhs_col_idx, num* sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(csr_lhs_row_idx, d_csr_lhs_row_idx, 5* sizeof(int), hipMemcpyDeviceToHost);

    cout << "csr value: \n";
    for(int i=0; i<num; i++){
        cout << csr_lhs[i] << "  ";
    }
    cout << endl;

    float alpha = 1.0f, beta = 0.0f;

    hipMemcpy(d_csr_lhs, host_csr_lhs, num* sizeof(float), hipMemcpyHostToDevice);

    hipsparseSpMatDescr_t matA;      // 稀疏矩阵的描述
    hipsparseDnMatDescr_t matB, matC;        // 稠密矩阵的描述
    void *dBuffer = nullptr;
    size_t bufferSize = 0;
    hipsparseSpSMDescr_t spSmDescr;      // 稀疏 * 稠密 的描述
    // 创建csr左矩阵
    CHECK_SPARSE(hipsparseCreateCsr(&matA, 4, 4, num, csr_lhs_row_idx, csr_lhs_col_idx, csr_lhs, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F));
    // 创建稠密的右矩阵和输出矩阵
    CHECK_SPARSE(hipsparseCreateDnMat(&matB, 4, 4, 4, d_rhs, HIP_R_32F, HIPSPARSE_ORDER_COL));
    CHECK_SPARSE(hipsparseCreateDnMat(&matC, 4, 4, 4, d_res, HIP_R_32F, HIPSPARSE_ORDER_COL));
    // 创建 spSM
    CHECK_SPARSE(hipsparseSpSM_createDescr(&spSmDescr));

    // 给稀疏矩阵添加填充属性
    hipsparseFillMode_t  fillmode = HIPSPARSE_FILL_MODE_LOWER;    // 以下三角填充？
    hipsparseSpMatSetAttribute(matA, HIPSPARSE_SPMAT_FILL_MODE, &fillmode, sizeof(fillmode));
    // 给稀疏矩阵添加对角线属性
    hipsparseDiagType_t  diagtype = HIPSPARSE_DIAG_TYPE_NON_UNIT;
    hipsparseSpMatSetAttribute(matA, HIPSPARSE_SPMAT_DIAG_TYPE, &diagtype, sizeof(diagtype));

    cout << "start cal: " << endl;
    CHECK_SPARSE(hipsparseSpSM_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, matB, matC, HIP_R_32F, HIPSPARSE_SPSM_ALG_DEFAULT, spSmDescr, &bufferSize));

    CHECK_SPARSE(hipsparseSpSM_analysis(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, matB, matC, HIP_R_32F, HIPSPARSE_SPSM_ALG_DEFAULT, spSmDescr, dBuffer));
    cout<< "middle cal: " << endl;
    CHECK_SPARSE(hipsparseSpSM_solve(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, matB, matC, HIP_R_32F, HIPSPARSE_SPSM_ALG_DEFAULT, spSmDescr));
    cout << "end cal: " << endl;

    CHECK_SPARSE(hipsparseDestroySpMat(matA));

    hipsparseDestroyDnMat(matB);
    hipsparseDestroyDnMat(matC);
    hipsparseSpSM_destroyDescr(spSmDescr);

    hipsparseDestroy(handle);


    hipMemcpy(res, d_res, 16* sizeof(float), hipMemcpyDeviceToHost);

    for(int i=0; i<16; i++){
        cout << res[i] << endl;
    }

    hipFree(d_lhs);
    hipFree(d_rhs);
    hipFree(d_csr_lhs);
    hipFree(d_csr_lhs_row_idx);
    hipFree(d_csr_lhs_col_idx);

    delete csr_lhs, csr_lhs_col_idx, csr_lhs_row_idx;
    delete[] nnzPerRowColumn;
}

int main(){
    // Host problem definition
    const int A_num_rows      = 4;
    const int A_num_cols      = 4;
    const int A_nnz           = 9;
    const int nrhs            = 2;
    const int ldb             = A_num_cols;
    const int ldc             = A_num_rows;
    int       hA_csrOffsets[] = { 0, 3, 4, 7, 9 };
    int       hA_columns[]    = { 0, 2, 3, 1, 0, 2, 3, 1, 3 };
    float     hA_values[]     = { 1.0f, 2.0f, 3.0f, 4.0f, 5.0f,
                                  6.0f, 7.0f, 8.0f, 9.0f };
    float     hB[]            = { 1.0f, 8.0f, 23.0f, 52.0f,
                                  1.0f, 8.0f, 23.0f, 52.0f };
    float     hC[]            = { 0.0f, 0.0f, 0.0f, 0.0f,
                                  0.0f, 0.0f, 0.0f, 0.0f };
    float     hY_result[]     = { 1.0f, 2.0f, 3.0f, 4.0f,
                                  1.0f, 2.0f, 3.0f, 4.0f };
    float     alpha           = 1.0f;
    //--------------------------------------------------------------------------
    // Device memory management
    int   *dA_csrOffsets, *dA_columns;
    float *dA_values, *dB, *dC;
    CHECK( hipMalloc((void**) &dA_csrOffsets,
                           (A_num_rows + 1) * sizeof(int)) );
    CHECK( hipMalloc((void**) &dA_columns, A_nnz * sizeof(int))       );
    CHECK( hipMalloc((void**) &dA_values,  A_nnz * sizeof(float))     );
    CHECK( hipMalloc((void**) &dB, nrhs * A_num_cols * sizeof(float)) );
    CHECK( hipMalloc((void**) &dC, nrhs * A_num_rows * sizeof(float)) );

    CHECK( hipMemcpy(dA_csrOffsets, hA_csrOffsets,
                           (A_num_rows + 1) * sizeof(int),
                           hipMemcpyHostToDevice) );
    CHECK( hipMemcpy(dA_columns, hA_columns, A_nnz * sizeof(int),
                           hipMemcpyHostToDevice) );
    CHECK( hipMemcpy(dA_values, hA_values, A_nnz * sizeof(float),
                           hipMemcpyHostToDevice) );
    CHECK( hipMemcpy(dB, hB, nrhs * A_num_cols * sizeof(float),
                           hipMemcpyHostToDevice) );
    CHECK( hipMemcpy(dC, hC, nrhs * A_num_rows * sizeof(float),
                           hipMemcpyHostToDevice) );
    //--------------------------------------------------------------------------
    // CUSPARSE APIs
    hipsparseHandle_t     handle = NULL;
    hipsparseSpMatDescr_t matA;
    hipsparseDnMatDescr_t matB, matC;
    void*                dBuffer    = NULL;
    size_t               bufferSize = 0;
    hipsparseSpSMDescr_t  spsmDescr;
    CHECK_SPARSE( hipsparseCreate(&handle) );
    // Create sparse matrix A in CSR format
    CHECK_SPARSE( hipsparseCreateCsr(&matA, A_num_rows, A_num_cols, A_nnz,
                                      dA_csrOffsets, dA_columns, dA_values,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F) );
    // Create dense vector X
    CHECK_SPARSE( hipsparseCreateDnMat(&matB, A_num_cols, nrhs, ldb, dB,
                                        HIP_R_32F, HIPSPARSE_ORDER_COL) );
    // Create dense vector y
    CHECK_SPARSE( hipsparseCreateDnMat(&matC, A_num_rows, nrhs, ldc, dC,
                                        HIP_R_32F, HIPSPARSE_ORDER_COL) );
    // Create opaque data structure, that holds analysis data between calls.
    CHECK_SPARSE( hipsparseSpSM_createDescr(&spsmDescr) );
    // Specify Lower|Upper fill mode.
    hipsparseFillMode_t fillmode = HIPSPARSE_FILL_MODE_LOWER;
    CHECK_SPARSE( hipsparseSpMatSetAttribute(matA, HIPSPARSE_SPMAT_FILL_MODE,
                                              &fillmode, sizeof(fillmode)) );
    // Specify Unit|Non-Unit diagonal type.
    hipsparseDiagType_t diagtype = HIPSPARSE_DIAG_TYPE_NON_UNIT;
    CHECK_SPARSE( hipsparseSpMatSetAttribute(matA, HIPSPARSE_SPMAT_DIAG_TYPE,
                                              &diagtype, sizeof(diagtype)) );
    // allocate an external buffer for analysis
    CHECK_SPARSE( hipsparseSpSM_bufferSize(
            handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
            HIPSPARSE_OPERATION_NON_TRANSPOSE,
            &alpha, matA, matB, matC, HIP_R_32F,
            HIPSPARSE_SPSM_ALG_DEFAULT, spsmDescr,
            &bufferSize) );
    CHECK( hipMalloc(&dBuffer, bufferSize) );
    CHECK_SPARSE( hipsparseSpSM_analysis(
            handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
            HIPSPARSE_OPERATION_NON_TRANSPOSE,
            &alpha, matA, matB, matC, HIP_R_32F,
            HIPSPARSE_SPSM_ALG_DEFAULT, spsmDescr, dBuffer) );
    // execute SpSM
    CHECK_SPARSE( hipsparseSpSM_solve(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                       HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                       &alpha, matA, matB, matC, HIP_R_32F,
                                       HIPSPARSE_SPSM_ALG_DEFAULT, spsmDescr) );

    // destroy matrix/vector descriptors
    CHECK_SPARSE( hipsparseDestroySpMat(matA) );
    CHECK_SPARSE( hipsparseDestroyDnMat(matB) );
    CHECK_SPARSE( hipsparseDestroyDnMat(matC) );
    CHECK_SPARSE( hipsparseSpSM_destroyDescr(spsmDescr));
    CHECK_SPARSE( hipsparseDestroy(handle) );
    //--------------------------------------------------------------------------
    // device result check
    CHECK( hipMemcpy(hC, dC, nrhs * A_num_rows * sizeof(float),
                           hipMemcpyDeviceToHost) );

    for (int i = 0; i < nrhs * A_num_rows; i++) {
        cout << hC[i] << endl;
    }

    //--------------------------------------------------------------------------
    // device memory deallocation
    CHECK( hipFree(dBuffer) );
    CHECK( hipFree(dA_csrOffsets) );
    CHECK( hipFree(dA_columns) );
    CHECK( hipFree(dA_values) );
    CHECK( hipFree(dB) );
    CHECK( hipFree(dC) );
    return 0;
}