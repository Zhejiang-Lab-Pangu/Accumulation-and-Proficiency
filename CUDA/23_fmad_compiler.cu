//
// Created by zjlab on 9/26/23.
//


#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * 使用 nvcc 的 --fmad 指令对编译进行优化
 *
 * */


__global__ void foo(float *ptr){

    *ptr = (*ptr) * (*ptr) + (*ptr);

}