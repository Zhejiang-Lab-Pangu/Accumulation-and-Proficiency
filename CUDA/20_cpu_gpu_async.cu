#include "hip/hip_runtime.h"
//
// Created by zjlab on 9/21/23.
//

#include <hip/hip_runtime.h>
#include <stdio.h>
#include "util.h"
using namespace std;


#define N_REPEAT 10
#define N_SEGMENT 1

void sumArrays(float * a,float * b,float * res,const int size)
{
    for(int i=0;i<size;i+=4)
    {
        res[i]=a[i]+b[i];
        res[i+1]=a[i+1]+b[i+1];
        res[i+2]=a[i+2]+b[i+2];
        res[i+3]=a[i+3]+b[i+3];
    }
}


__global__ void sumArraysGPU(float *a, float *b, float *res, const int N){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(idx<N){
        for(int j=0; j<N_REPEAT; j++){
            res[idx]  = a[idx] + b[idx];
        }
    }
}


void demo0(){
    initDevice(0);

    double iStart, iElaps;
    iStart = cpuSecond();
    int nElem = 1<< 24;
    printf("Vector size:%d\n",nElem);

    int nByte=sizeof(float)*nElem;

    float * a_h,*b_h,*res_h,*res_from_gpu_h;

    // 分配固定内存
    hipHostAlloc((float**)&a_h,nByte, hipHostMallocDefault);
    CHECK(hipHostAlloc((float**)&a_h,nByte, hipHostMallocDefault));
    CHECK(hipHostAlloc((float**)&b_h,nByte,hipHostMallocDefault));
    CHECK(hipHostAlloc((float**)&res_h,nByte,hipHostMallocDefault));
    CHECK(hipHostAlloc((float**)&res_from_gpu_h,nByte,hipHostMallocDefault));

    initialData(a_h,nElem);
    initialData(b_h,nElem);

    hipMemset(res_h, 0, nByte);
    hipMemset(res_from_gpu_h, 0, nByte);

    // cuda 内存
    float *a_d, *b_d, * res_d;
    CHECK(hipMalloc((float **)&a_d, nByte));
    CHECK(hipMalloc((float **)&b_d, nByte));
    CHECK(hipMalloc((float **)&res_d, nByte));


    //cpu
    sumArrays(a_h,b_h,res_h,nElem);


    // gpu
    dim3 block(512);
    dim3 grid((nElem-1)/block.x+1);

    int iElem=nElem/N_SEGMENT;      // 均匀切分 N_SEGMENT 块来计算
    hipStream_t stream[N_SEGMENT];
    for(int i=0;i<N_SEGMENT;i++)
    {
        CHECK(hipStreamCreate(&stream[i]));
    }

    hipEvent_t start,stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start,0);

    for (int i =0; i<N_SEGMENT; i++){
        int offset = i * iElem;

        CHECK(hipMemcpyAsync(&a_d[offset], &a_h[offset], nByte/N_SEGMENT, hipMemcpyHostToDevice, stream[i]));
        CHECK(hipMemcpyAsync(&b_d[offset], &b_h[offset], nByte/N_SEGMENT, hipMemcpyHostToDevice, stream[i]));

        sumArraysGPU<<<grid, block>>>(&a_d[offset], &b_d[offset], &res_d[offset], iElem);

        CHECK(hipMemcpyAsync(&res_from_gpu_h[offset], &res_d[offset], nByte/N_SEGMENT, hipMemcpyDeviceToHost, stream[i]));

    }


    // timer
    CHECK(hipEventRecord(stop, 0));
//    CHECK(hipEventSynchronize(stop));


    int counter = 0;
    while(hipEventQuery(stop) == hipErrorNotReady){
        counter ++;
    }
    printf("cpu counter:%d\n",counter);


    iElaps = cpuSecond() - iStart;
    printf("Asynchronous Execution configuration<<<%d,%d>>> Time elapsed %f sec\n",grid.x,block.x,iElaps);
    checkResult(res_h,res_from_gpu_h,nElem);


    for(int i=0;i<N_SEGMENT;i++)
    {
        CHECK(hipStreamDestroy(stream[i]));
    }
    hipFree(a_d);
    hipFree(b_d);
    hipFree(a_h);
    hipFree(b_h);
    hipFree(res_h);
    hipFree(res_from_gpu_h);
    hipEventDestroy(start);
    hipEventDestroy(stop);

}



int main(){

    demo0();

    return 0;
}