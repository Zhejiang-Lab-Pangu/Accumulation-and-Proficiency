#include "hip/hip_runtime.h"
//
// Created by zjlab on 9/12/23.
//
#include <iostream>
using namespace std;
#include <mma.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "util.h"

/*
 *
 *
 *  结构体数组 ： 并行编程范式，尤其是SIMD（单指令多数据）对SoA更友好。CUDA中普遍倾向于SoA因为这种内存访问可以有效地合并。
 *
 * */



// AoS的例子
struct naiveStruct{
    float a;
    float b;
};

void sumArrays(float * a,float * b,float * res,const int size) {

    for (int i = 0; i < size; i++) {
        res[i] = a[i] + b[i];
    }
}

__global__ void sumArraysGPU(float*a, float*b, struct naiveStruct* res, int n)
{
    //int i=threadIdx.x;
    int i= blockIdx.x * blockDim.x + threadIdx.x;
    if(i<n)
        res[i].a=a[i]+b[i];
}

void checkResult_struct(float* res_h, struct naiveStruct*res_from_gpu_h, int nElem)
{
    for(int i=0;i<nElem;i++)
        if (res_h[i]!=res_from_gpu_h[i].a)
        {
            printf("check fail!\n");
            exit(0);
        }
    printf("result check success!\n");
}


void demo0(int argc, char **argv){
    int dev = 0;
    hipSetDevice(dev);

    int nElem=1<<18;
    int offset=0;
    if(argc>=2)
        offset=atoi(argv[1]);
    printf("Vector size:%d\n",nElem);
    int nByte=sizeof(float)*nElem;
    int nByte_struct = sizeof(naiveStruct) * nElem;



    float *a_h=(float*)malloc(nByte);
    float *b_h=(float*)malloc(nByte);
    float *res_h=(float*)malloc(nByte);
    naiveStruct *res_from_gpu_h=(naiveStruct*)malloc(nByte_struct);
    memset(res_h,0,nByte);
    memset(res_from_gpu_h,0,nByte);


    float *a_d, *b_d;
    naiveStruct * res_d;
    CHECK(hipMalloc((float**)&a_d,nByte));
    CHECK(hipMalloc((float**)&b_d,nByte));
    CHECK(hipMalloc((float**)&res_d,nByte_struct));
    CHECK(hipMemset(res_d,0,nByte_struct));
    initialData(a_h,nElem);
    initialData(b_h,nElem);

    CHECK(hipMemcpy(a_d,a_h,nByte,hipMemcpyHostToDevice));
    CHECK(hipMemcpy(b_d,b_h,nByte,hipMemcpyHostToDevice));


    dim3 block(1024);
    dim3 grid(nElem/block.x);
    double iStart,iElaps;
    iStart=cpuSecond();
    sumArraysGPU<<<grid,block>>>(a_d, b_d, res_d, nElem);
    hipDeviceSynchronize();
    iElaps=cpuSecond()-iStart;
    CHECK(hipMemcpy(res_from_gpu_h,res_d,nByte_struct,hipMemcpyDeviceToHost));
    printf("Execution configuration<<<%d,%d>>> Time elapsed %f sec \n", grid.x, block.x, iElaps);

    sumArrays(a_h, b_h, res_h, nElem);

    checkResult_struct(res_h, res_from_gpu_h, nElem);
    hipFree(a_d);
    hipFree(b_d);
    hipFree(res_d);

    free(a_h);
    free(b_h);
    free(res_h);
    free(res_from_gpu_h);


}


void sumArrays(float * a,float * b,float * res,int offset,const int size)
{

    for(int i=0,k=offset;k<size;i++,k++)
    {
        res[i]=a[k]+b[k];
    }

}
__global__ void sumArraysGPU(float*a,float*b,float*res,int offset,int n)
{
    //int i=threadIdx.x;
    int i=blockIdx.x*blockDim.x*4+threadIdx.x;
    int k=i+offset;
    if(k+3*blockDim.x<n)
    {
        res[i]=a[k]+b[k];
        res[i+blockDim.x]=a[k+blockDim.x]+b[k+blockDim.x];
        res[i+blockDim.x*2]=a[k+blockDim.x*2]+b[k+blockDim.x*2];
        res[i+blockDim.x*3]=a[k+blockDim.x*3]+b[k+blockDim.x*3];
    }

}
void demo1(int argc, char **argv){
    int dev = 0;
    hipSetDevice(dev);
    int block_x=512;
    int nElem=1<<18;
    int offset=0;
    if(argc==2)
        offset=atoi(argv[1]);
    else if(argc==3)
    {
        offset=atoi(argv[1]);
        block_x=atoi(argv[2]);
    }
    printf("Vector size:%d\n",nElem);
    int nByte=sizeof(float)*nElem;
    float *a_h=(float*)malloc(nByte);
    float *b_h=(float*)malloc(nByte);
    float *res_h=(float*)malloc(nByte);
    float *res_from_gpu_h=(float*)malloc(nByte);
    memset(res_h,0,nByte);
    memset(res_from_gpu_h,0,nByte);

    float *a_d,*b_d,*res_d;
    CHECK(hipMalloc((float**)&a_d,nByte));
    CHECK(hipMalloc((float**)&b_d,nByte));
    CHECK(hipMalloc((float**)&res_d,nByte));
    CHECK(hipMemset(res_d,0,nByte));
    initialData(a_h,nElem);
    initialData(b_h,nElem);

    CHECK(hipMemcpy(a_d,a_h,nByte,hipMemcpyHostToDevice));
    CHECK(hipMemcpy(b_d,b_h,nByte,hipMemcpyHostToDevice));

    dim3 block(block_x);
    dim3 grid(nElem/block.x);
    double iStart,iElaps;
    iStart=cpuSecond();
    sumArraysGPU<<<grid,block>>>(a_d,b_d,res_d,offset,nElem);
    hipDeviceSynchronize();
    iElaps=cpuSecond()-iStart;

    printf("warmup Time elapsed %f sec\n",iElaps);
    iStart=cpuSecond();
    sumArraysGPU<<<grid,block>>>(a_d,b_d,res_d,offset,nElem);
    hipDeviceSynchronize();
    iElaps=cpuSecond()-iStart;
    CHECK(hipMemcpy(res_from_gpu_h,res_d,nByte,hipMemcpyDeviceToHost));
    printf("Execution configuration<<<%d,%d>>> Time elapsed %f sec --offset:%d \n",grid.x,block.x,iElaps,offset);


    sumArrays(a_h,b_h,res_h,offset,nElem);

    checkResult(res_h,res_from_gpu_h,nElem-4*block_x);
    hipFree(a_d);
    hipFree(b_d);
    hipFree(res_d);

    free(a_h);
    free(b_h);
    free(res_h);
    free(res_from_gpu_h);

}


int main(int argc, char **argv) {
//    demo0(argc, argv);
    demo1(argc, argv);
    return 0;
}