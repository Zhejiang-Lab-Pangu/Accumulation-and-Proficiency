
//
#include <iostream>
using namespace std;
#include <mma.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
//using namespace nvcuda;
#include <sys/time.h>


/*
 *
 * cpu计时
 *
 * 核函数计时：
 *      nvprof [nvprof_args] <application>[application_args]
 *
 * */

#define CHECK(call)\
{\
  const hipError_t error=call;\
  if(error!=hipSuccess)\
  {\
      printf("ERROR: %s:%d,",__FILE__,__LINE__);\
      printf("code:%d,reason:%s\n",error,hipGetErrorString(error));\
      exit(1);\
  }\
}

void initialData(float* ip,int size)
{
    time_t t;       // ime_t 这种类型就是用来存储从1970年到现在经过了多少秒
    srand((unsigned )time(&t));
    for(int i=0;i<size;i++)
    {
        ip[i]=(float)(rand()&0xffff)/1000.0f;   // 0xffff 2^16-1
    }
}

double cpuSecond(){
    timeval tp;
    gettimeofday(&tp, nullptr);     // 初始化tp：即返回现在距1970的时间

    return ((double)tp.tv_sec + (double)tp.tv_usec * 1e-6);
}


//void sumArrays(float * a,float * b,float * res,const int size)
//{
//    for(int i=0;i<size;i+=4)
//    {
//        res[i]=a[i]+b[i];
//        res[i+1]=a[i+1]+b[i+1];
//        res[i+2]=a[i+2]+b[i+2];
//        res[i+3]=a[i+3]+b[i+3];
//    }
//}

//void printRslt(float*p, int size){
//    for(int i=0; i<size; i++){
//        cout << p[i] << "  ";
//    }
//    cout << endl;
//}

// 线程如何组织：threadIdx(dim3) 和 blockIdx (dim3)
__global__ void sumArraysGPU(float * a,float * b,float * res){
    int idx = threadIdx.x + blockIdx.x * gridDim.x;
    res[idx] = a[idx] + b[idx];
}


void demo0(){
    int dev = 0;
    hipSetDevice(dev);

    int nElem= (1 << 16) + 1 ;
    printf("Vector size:%d\n",nElem);
    int nByte=sizeof(float) * nElem;
    float *a_h=(float*)malloc(nByte);
    float *b_h=(float*)malloc(nByte);
    float *res_h=(float*)malloc(nByte);
    float *res_from_gpu_h=(float*)malloc(nByte);
    memset(res_h,0,nByte);
    memset(res_from_gpu_h,0,nByte);

    float *a_d, *b_d, *res_d;
    CHECK(hipMalloc((void **)&a_d, nByte));
    CHECK(hipMalloc((void **)&b_d, nByte));
    CHECK(hipMalloc((void **)&res_d, nByte));

    initialData(a_h,nElem);
    initialData(b_h,nElem);

    CHECK(hipMemcpy(a_d,a_h,nByte,hipMemcpyHostToDevice));
    CHECK(hipMemcpy(b_d,b_h,nByte,hipMemcpyHostToDevice));

    unsigned int block_size = 1024;
    dim3 block(block_size);
    dim3 grid((nElem-1)/block.x+1);
    printf("grid.x %d block.x %d\n",grid.x,block.x);

    // 时间分析
    double iStart,iElaps;
    iStart=cpuSecond();
    sumArraysGPU<<<grid, block>>>(a_d, b_d, res_d);
    hipDeviceSynchronize();
    iElaps=cpuSecond()-iStart;
    printf("time cost: %f\n", iElaps);

    CHECK(hipMemcpy(res_from_gpu_h,res_d,nByte,hipMemcpyDeviceToHost));

//    sumArrays(a_h,b_h,res_h,nElem);

    hipFree(a_d);
    hipFree(b_d);
    hipFree(res_d);

    free(a_h);
    free(b_h);
    free(res_h);
    free(res_from_gpu_h);
}

int main()
{
    demo0();

    return 0;
}
