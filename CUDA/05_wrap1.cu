#include "hip/hip_runtime.h"
//
// Created by zjlab on 9/5/23.
//

#include <iostream>
using namespace std;
#include <mma.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include<math.h>
#include "util.h"


/*
 *
 * wrap1
 *
 * */


// warmup部分是提前启动一次GPU，因为第一次启动GPU时会比第二次速度慢一些
__global__ void warmup(float *c){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float a = 0.0, b= 0.0;

    if((tid / warpSize) % 2 == 0){
        a = 100.0f;
    } else{
        b = 200.0f;
    }

    c[tid] = a + b;
}

__global__ void mathKernel1(float *c){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float a = 0.0, b= 0.0;

    if(tid % 2 == 0){
        a = 100.0f;
    } else{
        b = 200.0f;
    }

    c[tid] = a + b;
}

__global__ void mathKernel2(float *c)
{
    int tid = blockIdx.x* blockDim.x + threadIdx.x;
    float a = 0.0;
    float b = 0.0;
    if ((tid/warpSize) % 2 == 0)
    {
        a = 100.0f;
    }
    else
    {
        b = 200.0f;
    }
    c[tid] = a + b;
}

__global__ void mathKernel3(float *c)
{
    int tid = blockIdx.x* blockDim.x + threadIdx.x;
    float a = 0.0;
    float b = 0.0;
    bool ipred = (tid % 2 == 0);
    if (ipred)
    {
        a = 100.0f;
    }
    else
    {
        b = 200.0f;
    }
    c[tid] = a + b;
}

int main(int argc, char **argv){
    int dev = 0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    printf("%s using Device %d: %s\n", argv[0], dev, deviceProp.name);

    //set up data size
    int size = 64;
    int blocksize = 64;
    if (argc > 1) blocksize = atoi(argv[1]);
    if (argc > 2) size = atoi(argv[2]);
    printf("Data size %d ", size);

    //set up execution configuration
    dim3 block(blocksize,1);
    dim3 grid((size - 1) / block.x + 1,1);
    printf("Execution Configure (block %d grid %d)\n", block.x, grid.x);

    //allocate memory
    float * C_dev;
    size_t nBytes = size * sizeof(float);
    float * C_host=(float*)malloc(nBytes);
    hipMalloc((float**)&C_dev, nBytes);

    //run a warmup kernel to remove overhead
    double iStart, iElaps;
    hipDeviceSynchronize();
    iStart = cpuSecond();
    warmup<<<grid,block>>> (C_dev);
    hipDeviceSynchronize();
    iElaps = cpuSecond() - iStart;
    printf("warmup	  <<<%d,%d>>>elapsed %lf sec \n", grid.x, block.x, iElaps);

    //run kernel 1
    iStart = cpuSecond();
    mathKernel1 <<< grid,block >>> (C_dev);
    hipDeviceSynchronize();
    iElaps = cpuSecond() - iStart;
    printf("mathKernel1<<<%4d,%4d>>>elapsed %lf sec \n", grid.x, block.x, iElaps);
    hipMemcpy(C_host,C_dev,nBytes,hipMemcpyDeviceToHost);

    //run kernel 2
    iStart = cpuSecond();
    mathKernel2 <<<grid,block >>> (C_dev);
    hipDeviceSynchronize();
    iElaps = cpuSecond() - iStart;
    printf("mathKernel2<<<%4d,%4d>>>elapsed %lf sec \n", grid.x, block.x, iElaps);

    //run kernel 3
    iStart = cpuSecond();
    mathKernel3 << <grid, block >> > (C_dev);
    hipDeviceSynchronize();
    iElaps = cpuSecond() - iStart;
    printf("mathKernel3<<<%4d,%4d>>>elapsed %lf sec \n", grid.x, block.x, iElaps);

    hipFree(C_dev);
    free(C_host);
    hipDeviceReset();
    return EXIT_SUCCESS;

}




