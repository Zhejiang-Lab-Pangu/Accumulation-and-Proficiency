//
// Created by root on 8/30/23.
//
#include <iostream>
using namespace std;
#include <mma.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
//using namespace nvcuda;


/*
 * check dim
 * 线程的组织形式： dim3
 *
 * i = threadIdx + gridDim * blockIdx;
 *
 * */


// 线程如何组织：threadIdx(dim3) 和 blockIdx (dim3)
__global__ void checkIndex(void){
    printf("threadIdx:(%d,%d,%d) blockIdx:(%d,%d,%d) blockDim:(%d,%d,%d)  gridDim(%d,%d,%d)\n",
           threadIdx.x,threadIdx.y,threadIdx.z,
           blockIdx.x,blockIdx.y,blockIdx.z,
           blockDim.x,blockDim.y,blockDim.z,
           gridDim.x,gridDim.y,gridDim.z);
}

void demo0(){
    int nElem=6;
    dim3 block(3, 3);
    dim3 grid((nElem+block.x-1)/block.x, (nElem+block.y-1)/block.y);
    printf("grid.x %d grid.y %d grid.z %d\n",grid.x,grid.y,grid.z);
    printf("block.x %d block.y %d block.z %d\n",block.x,block.y,block.z);
    checkIndex <<<grid,block>>> ();
    hipDeviceReset();

}
void demo1(){
    int nElem=1024;
    dim3 block(1024);
    dim3 grid((nElem-1)/block.x+1);
    printf("grid.x %d block.x %d\n",grid.x,block.x);

    block.x=512;
    grid.x=(nElem-1)/block.x+1;
    printf("grid.x %d block.x %d\n",grid.x,block.x);

    block.x=256;
    grid.x=(nElem-1)/block.x+1;
    printf("grid.x %d block.x %d\n",grid.x,block.x);

    block.x=128;
    grid.x=(nElem-1)/block.x+1;
    printf("grid.x %d block.x %d\n",grid.x,block.x);

//    cudaDeviceReset();
}

int main()
{
    demo0();
    cout << endl;
    demo1();
    return 0;
}
