#include "hip/hip_runtime.h"
//
// Created by zjlab on 9/12/23.
//
#include <iostream>
using namespace std;
#include <mma.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "util.h"

/*
 * CUDA 内存对齐
 *
 * nvcc 指令
 *      (1) nvcc  -Xptxas -dlcm=cg     禁用一级缓存
 *      (2) nvcc  -Xptxas -dlcm=ca     启用一级缓存
 *
 *
 *  结构体数组 ： 并行编程范式，尤其是SIMD（单指令多数据）对SoA更友好。CUDA中普遍倾向于SoA因为这种内存访问可以有效地合并。
 *
 * */

void sumArrays(float * a,float * b,float * res,int offset,const int size)
{

    for(int i=0,k=offset;k<size;i++,k++)
    {
        res[i]=a[k]+b[k];
    }

}

__global__ void sumArraysGPU(float*a, float*b, float*res, int offset, int n)
{
    //int i=threadIdx.x;
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int k = i + offset;
    if(k<n)
        res[i]=a[k]+b[k];
}


struct naiveStruct{
    float a;
    float b;
};



// hipMalloc
void demo0(int argc, char **argv){
    int dev = 0;
    hipSetDevice(dev);

    int nElem=1<<18;
    int offset=0;
    if(argc>=2)
        offset=atoi(argv[1]);
    printf("Vector size:%d\n",nElem);
    int nByte=sizeof(float)*nElem;

    float *a_h=(float*)malloc(nByte);
    float *b_h=(float*)malloc(nByte);
    float *res_h=(float*)malloc(nByte);
    float *res_from_gpu_h=(float*)malloc(nByte);
    memset(res_h,0,nByte);
    memset(res_from_gpu_h,0,nByte);


    float *a_d,*b_d,*res_d;
    CHECK(hipMalloc((float**)&a_d,nByte));
    CHECK(hipMalloc((float**)&b_d,nByte));
    CHECK(hipMalloc((float**)&res_d,nByte));
    CHECK(hipMemset(res_d,0,nByte));
    initialData(a_h,nElem);
    initialData(b_h,nElem);

    CHECK(hipMemcpy(a_d,a_h,nByte,hipMemcpyHostToDevice));
    CHECK(hipMemcpy(b_d,b_h,nByte,hipMemcpyHostToDevice));


    dim3 block(1024);
    dim3 grid(nElem/block.x);
    double iStart,iElaps;
    iStart=cpuSecond();
    sumArraysGPU<<<grid,block>>>(a_d,b_d,res_d, offset,nElem);
    hipDeviceSynchronize();
    iElaps=cpuSecond()-iStart;
    CHECK(hipMemcpy(res_from_gpu_h,res_d,nByte,hipMemcpyDeviceToHost));
    printf("Execution configuration<<<%d,%d>>> Time elapsed %f sec --offset:%d \n", grid.x, block.x, iElaps, offset);

    sumArrays(a_h,b_h,res_h,offset,nElem);

    checkResult(res_h,res_from_gpu_h,nElem);
    hipFree(a_d);
    hipFree(b_d);
    hipFree(res_d);

    free(a_h);
    free(b_h);
    free(res_h);
    free(res_from_gpu_h);


}




int main(int argc, char **argv) {
    demo0(argc, argv);
//    cout << endl;
//    demo1();
    return 0;
}