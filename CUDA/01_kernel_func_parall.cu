
//
#include <iostream>
using namespace std;
#include <mma.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
//using namespace nvcuda;


/*
 * 1. 核函数对串行代码块for的操作
 * kernel func: for -> parallel
 *
 * 2.cpu计时
 *
 * */

#define CHECK(call)\
{\
  const hipError_t error=call;\
  if(error!=hipSuccess)\
  {\
      printf("ERROR: %s:%d,",__FILE__,__LINE__);\
      printf("code:%d,reason:%s\n",error,hipGetErrorString(error));\
      exit(1);\
  }\
}

void initialData(float* ip,int size)
{
    time_t t;       // ime_t 这种类型就是用来存储从1970年到现在经过了多少秒
    srand((unsigned )time(&t));
    for(int i=0;i<size;i++)
    {
        ip[i]=(float)(rand()&0xffff)/1000.0f;   // 0xffff 2^16-1
    }
}



void sumArrays(float * a,float * b,float * res,const int size)
{
    for(int i=0;i<size;i+=4)
    {
        res[i]=a[i]+b[i];
        res[i+1]=a[i+1]+b[i+1];
        res[i+2]=a[i+2]+b[i+2];
        res[i+3]=a[i+3]+b[i+3];
    }
}

void printRslt(float*p, int size){
    for(int i=0; i<size; i++){
        cout << p[i] << "  ";
    }
    cout << endl;
}

// 线程如何组织：threadIdx(dim3) 和 blockIdx (dim3)
__global__ void sumArraysGPU(float * a,float * b,float * res){
    int idx = threadIdx.x + blockIdx.x * gridDim.x;
    res[idx] = a[idx] + b[idx];
}


void demo0(){
    int dev = 0;
    hipSetDevice(dev);

    int nElem=32;
    printf("Vector size:%d\n",nElem);
    int nByte=sizeof(float) * nElem;
    float *a_h=(float*)malloc(nByte);
    float *b_h=(float*)malloc(nByte);
    float *res_h=(float*)malloc(nByte);
    float *res_from_gpu_h=(float*)malloc(nByte);
    memset(res_h,0,nByte);
    memset(res_from_gpu_h,0,nByte);

    float *a_d, *b_d, *res_d;
    CHECK(hipMalloc((void **)&a_d, nByte));
    CHECK(hipMalloc((void **)&b_d, nByte));
    CHECK(hipMalloc((void **)&res_d, nByte));

    initialData(a_h,nElem);
    initialData(b_h,nElem);

    CHECK(hipMemcpy(a_d,a_h,nByte,hipMemcpyHostToDevice));
    CHECK(hipMemcpy(b_d,b_h,nByte,hipMemcpyHostToDevice));

    dim3 block(nElem);
    dim3 grid((nElem-1)/block.x+1);
    printf("grid.x %d block.x %d\n",grid.x,block.x);
    sumArraysGPU<<<grid, block>>>(a_d, b_d, res_d);

    CHECK(hipMemcpy(res_from_gpu_h,res_d,nByte,hipMemcpyDeviceToHost));       // 隐式方法等待设备执行完成

    sumArrays(a_h,b_h,res_h,nElem);

    printRslt(res_h, nElem);
    printRslt(res_from_gpu_h, nElem);

    hipFree(a_d);
    hipFree(b_d);
    hipFree(res_d);

    free(a_h);
    free(b_h);
    free(res_h);
    free(res_from_gpu_h);
}

int main()
{
    demo0();

    return 0;
}
