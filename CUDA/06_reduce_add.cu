#include "hip/hip_runtime.h"
//
// Created by zjlab on 9/7/23.
//
#include <iostream>
using namespace std;
#include <mma.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include<math.h>
#include "util.h"


/*
 *
 * reduce add
 *
 * */


__global__ void warmup(int * g_idata, int * g_odata, unsigned int n)
{
    //set thread ID
    unsigned int tid = threadIdx.x;
    //boundary check
    if (tid >= n) return;
    //convert global data pointer to the
    int *idata = g_idata + blockIdx.x*blockDim.x;
    //in-place reduction in global memory
    for (int stride = 1; stride < blockDim.x; stride *= 2)
    {
        if ((tid % (2 * stride)) == 0)
        {
            idata[tid] += idata[tid + stride];
        }
        //synchronize within block
        __syncthreads();
    }
    //write result for this block to global mem
    if (tid == 0)
        g_odata[blockIdx.x] = idata[0];

}


__global__ void reduceNeighbored(int * g_idata,int * g_odata,unsigned int n)
{
    unsigned int tid = threadIdx.x;

    if(tid>n) return;

    int *idata = g_idata+ blockDim.x * blockIdx.x;

    for(int stride=1; stride<blockDim.x; stride*=2){
        if((tid % (2 * stride)) == 0){
            idata[tid] += idata[tid + stride];
        }
        __syncthreads();        //synchronize within block  每轮操作都是并行的，不保证所有线程能同时执行完毕，所以需要等待；避免块内的线程竞争内存
    }

    if(tid == 0){
        g_odata[blockIdx.x] = idata[0];
    }
}


__global__ void reduceNeighboredLess(int * g_idata,int *g_odata,unsigned int n)
{
    unsigned int tid = threadIdx.x;
    unsigned idx = blockIdx.x*blockDim.x + threadIdx.x;
    // convert global data pointer to the local point of this block
    int *idata = g_idata + blockIdx.x*blockDim.x;
    if (idx > n)
        return;
    //in-place reduction in global memory
    for (int stride = 1; stride < blockDim.x; stride *= 2)
    {
        //convert tid into local array index
        int index = 2 * stride *tid;
        if (index < blockDim.x)
        {
            idata[index] += idata[index + stride];
        }
        __syncthreads();
    }
    //write result for this block to global men
    if (tid == 0)
        g_odata[blockIdx.x] = idata[0];
}


__global__ void reduceInterleaved(int * g_idata, int *g_odata, unsigned int n)
{
    unsigned int tid = threadIdx.x;
    unsigned idx = blockIdx.x*blockDim.x + threadIdx.x;
    // convert global data pointer to the local point of this block
    int *idata = g_idata + blockIdx.x*blockDim.x;
    if (idx >= n)
        return;
    //in-place reduction in global memory
    for (int stride = blockDim.x/2; stride >0; stride >>=1)
    {

        if (tid <stride)
        {
            idata[tid] += idata[tid + stride];
        }
        __syncthreads();
    }
    //write result for this block to global men
    if (tid == 0)
        g_odata[blockIdx.x] = idata[0];
}


// 展开的规约 - 展开尺度2
__global__ void reduceUnroll2(int *g_idata, int *g_odata, unsigned int n){

    unsigned int tid = threadIdx.x;
    unsigned int idx = blockDim.x * blockIdx.x * 2 + threadIdx.x;

    if(tid>=n) return;

    int *idata = g_idata + blockIdx.x * blockDim.x *2;

    if(idx+blockDim.x < n){
        g_idata[idx] += g_idata[idx+blockDim.x];
    }
    __syncthreads();

    for(int stride = blockDim.x/2; stride>0; stride>>=1){
        if(tid < stride){
            idata[tid] += idata[tid+stride];
        }
        __syncthreads();
    }

    if(tid == 0){
        g_odata[blockIdx.x] = idata[0];
    }
}



// 展开的规约 - 展开尺度4
__global__ void reduceUnroll4(int *g_idata, int *g_odata, unsigned  int n){

    unsigned int tid = threadIdx.x;
    unsigned int idx = blockDim.x * blockIdx.x * 4 + threadIdx.x;

    if(tid >=n) return;

    int *idata = g_idata + blockDim.x * blockIdx.x * 4;

    if(idx + 3 * blockDim.x < n){
        g_idata[idx] += g_idata[idx+blockDim.x];
        g_idata[idx] += g_idata[idx+blockDim.x * 2];
        g_idata[idx] += g_idata[idx+blockDim.x * 3];
    }
    __syncthreads();

    for(int stride = blockDim.x/2; stride>0; stride>>=1){
        if(tid < stride){
            idata[tid] += idata[tid+stride];
        }
        __syncthreads();
    }

    if(tid == 0){
        g_odata[blockIdx.x] = idata[0];
    }
}


// 展开的规约 - 展开尺度8
__global__ void reduceUnroll8(int *g_idata, int *g_odata, unsigned  int n){

    unsigned int tid = threadIdx.x;
    unsigned int idx = blockDim.x * blockIdx.x * 8 + threadIdx.x;

    if(tid >=n) return;

    int *idata = g_idata + blockDim.x * blockIdx.x * 8;

    if(idx + 7 * blockDim.x < n){
        g_idata[idx] += g_idata[idx+blockDim.x];
        g_idata[idx] += g_idata[idx+blockDim.x * 2];
        g_idata[idx] += g_idata[idx+blockDim.x * 3];
        g_idata[idx] += g_idata[idx+blockDim.x * 4];
        g_idata[idx] += g_idata[idx+blockDim.x * 5];
        g_idata[idx] += g_idata[idx+blockDim.x * 6];
        g_idata[idx] += g_idata[idx+blockDim.x * 7];
    }
    __syncthreads();

    for(int stride = blockDim.x/2; stride>0; stride>>=1){
        if(tid < stride){
            idata[tid] += idata[tid+stride];
        }
        __syncthreads();
    }

    if(tid == 0){
        g_odata[blockIdx.x] = idata[0];
    }
}



// 完全展开的规约 - 在warp级别中
__global__ void reduceUnrollWarp8(int *g_idata, int *g_odata, unsigned  int n){

    unsigned int tid = threadIdx.x;
    unsigned int idx = blockDim.x * blockIdx.x * 8 + threadIdx.x;

    if(tid >=n) return;

    int *idata = g_idata + blockDim.x * blockIdx.x * 8;

    if(idx + 7 * blockDim.x < n){
        g_idata[idx] += g_idata[idx+blockDim.x];
        g_idata[idx] += g_idata[idx+blockDim.x * 2];
        g_idata[idx] += g_idata[idx+blockDim.x * 3];
        g_idata[idx] += g_idata[idx+blockDim.x * 4];
        g_idata[idx] += g_idata[idx+blockDim.x * 5];
        g_idata[idx] += g_idata[idx+blockDim.x * 6];
        g_idata[idx] += g_idata[idx+blockDim.x * 7];
    }
    __syncthreads();

    for(int stride = blockDim.x/2; stride>32; stride>>=1){
        if(tid < stride){
            idata[tid] += idata[tid+stride];
        }
        __syncthreads();
    }

    if(tid < 32){

        // 添加volatile，防止编译器优化数据传输而打乱执行顺序
        volatile int *vmem = idata;     // 控制变量结果写回到内存，而不是存在共享内存，或者缓存中;

        vmem[tid] += vmem[tid+32];      // 当执行 tid+32的时候，这32个线程都在执行这步,不会进入到下一句，以下同理
        vmem[tid] += vmem[tid+16];
        vmem[tid] += vmem[tid+8];
        vmem[tid] += vmem[tid+4];
        vmem[tid] += vmem[tid+2];
        vmem[tid] += vmem[tid+1];
    }

    if(tid == 0){
        g_odata[blockIdx.x] = idata[0];
    }
}



// 模板函数的归约
__global__ void reduceCompleteUnrollWarp8(int * g_idata,int * g_odata,unsigned int n)
{
    //set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockDim.x*blockIdx.x*8+threadIdx.x;
    //boundary check
    if (tid >= n) return;
    //convert global data pointer to the
    int *idata = g_idata + blockIdx.x*blockDim.x*8;
    if(idx+7 * blockDim.x<n)
    {
        int a1=g_idata[idx];
        int a2=g_idata[idx+blockDim.x];
        int a3=g_idata[idx+2*blockDim.x];
        int a4=g_idata[idx+3*blockDim.x];
        int a5=g_idata[idx+4*blockDim.x];
        int a6=g_idata[idx+5*blockDim.x];
        int a7=g_idata[idx+6*blockDim.x];
        int a8=g_idata[idx+7*blockDim.x];
        g_idata[idx]=a1+a2+a3+a4+a5+a6+a7+a8;

    }
    __syncthreads();

    //in-place reduction in global memory
//    printf("blockdim: %d\n", blockDim.x);
    if(blockDim.x>=1024 && tid <512)
        idata[tid]+=idata[tid+512];
    __syncthreads();

//    printf("blockdim: %d\n", blockDim.x);
    if(blockDim.x>=512 && tid <256)
        idata[tid]+=idata[tid+256];
    __syncthreads();

    if(blockDim.x>=256 && tid <128)
        idata[tid]+=idata[tid+128];
    __syncthreads();

    if(blockDim.x>=128 && tid <64)
        idata[tid]+=idata[tid+64];
    __syncthreads();

    //write result for this block to global mem
    if(tid<32)
    {
        volatile int *vmem = idata;
        vmem[tid]+=vmem[tid+32];
        vmem[tid]+=vmem[tid+16];
        vmem[tid]+=vmem[tid+8];
        vmem[tid]+=vmem[tid+4];
        vmem[tid]+=vmem[tid+2];
        vmem[tid]+=vmem[tid+1];

    }

    if (tid == 0)
        g_odata[blockIdx.x] = idata[0];

}




int main(int argc, char **argv){
    int dev = 0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    printf("%s using Device %d: %s\n", argv[0], dev, deviceProp.name);

    //set up data size
    int size = 1 << 24;
    printf("Data size %d ", size);
    int blocksize = 1024;
    if (argc > 1)
    {
        blocksize = atoi(argv[1]);
    }


    //set up execution configuration
    dim3 block(blocksize,1);
    dim3 grid((size - 1) / block.x + 1,1);
    printf("Execution Configure (block %d grid %d)\n", block.x, grid.x);

    //allocate host memory
    size_t nBytes = size * sizeof(int);
    int * idata_host=(int*)malloc(nBytes);
    int *odata_host = (int*) malloc(grid.x * sizeof(int));
    int * tmp = (int*)malloc(nBytes);

    //initialize the array
    initialData_int(idata_host, size);
    memcpy(tmp, idata_host, nBytes);

//    hipMalloc((float**)&C_dev, nBytes);
    double iStart, iElaps;
    int gpu_sum = 0;


    // device memory
    int * idata_dev = nullptr;
    int * odata_dev = nullptr;
    CHECK(hipMalloc((void**)&idata_dev, nBytes));
    CHECK(hipMalloc((void**)&odata_dev, grid.x * sizeof(int)));


    //cpu reduction
    int cpu_sum = 0;
    iStart = cpuSecond();
    //cpu_sum = recursiveReduce(tmp, size);
    for (int i = 0; i < size; i++)
        cpu_sum += tmp[i];
    printf("cpu sum:%d \n", cpu_sum);
    iElaps = cpuSecond() - iStart;
    printf("cpu reduce                 elapsed %lf ms cpu_sum: %d\n", iElaps, cpu_sum);


    //cpu reduction1
    iStart = cpuSecond();
    //cpu_sum = recursiveReduce(tmp, size);
    cpu_sum = recursiveReduce(tmp, size);
    printf("cpu sum:%d \n", cpu_sum);
    iElaps = cpuSecond() - iStart;
    printf("cpu recursiveReduce        elapsed %lf ms cpu_sum: %d\n", iElaps, cpu_sum);

    // warmup
    CHECK(hipMemcpy(idata_dev, idata_host, nBytes, hipMemcpyHostToDevice));
    CHECK(hipDeviceSynchronize());
    iStart = cpuSecond();
    warmup <<<grid, block >>>(idata_dev, odata_dev, size);
    hipDeviceSynchronize();
    iElaps = cpuSecond() - iStart;
    hipMemcpy(odata_host, odata_dev, grid.x * sizeof(int), hipMemcpyDeviceToHost);
    gpu_sum = 0;
    for (int i = 0; i < grid.x; i++)
        gpu_sum += odata_host[i];
    printf("gpu warmup                 elapsed %lf ms gpu_sum: %d<<<grid %d block %d>>>\n", iElaps, gpu_sum, grid.x, block.x);

    //kernel 1:reduceNeighbored

    CHECK(hipMemcpy(idata_dev, idata_host, nBytes, hipMemcpyHostToDevice));
    CHECK(hipDeviceSynchronize());
    iStart = cpuSecond();
    reduceNeighbored <<<grid, block >>>(idata_dev, odata_dev, size);
    hipDeviceSynchronize();
    iElaps = cpuSecond() - iStart;
    hipMemcpy(odata_host, odata_dev, grid.x * sizeof(int), hipMemcpyDeviceToHost);
    gpu_sum = 0;
    for (int i = 0; i < grid.x; i++)
        gpu_sum += odata_host[i];
    printf("gpu reduceNeighbored       elapsed %lf ms gpu_sum: %d<<<grid %d block %d>>>\n", iElaps, gpu_sum, grid.x, block.x);

    //kernel 2:reduceNeighboredLess

    CHECK(hipMemcpy(idata_dev, idata_host, nBytes, hipMemcpyHostToDevice));
    CHECK(hipDeviceSynchronize());
    iStart = cpuSecond();
    reduceNeighboredLess <<<grid, block>>>(idata_dev, odata_dev, size);
    hipDeviceSynchronize();
    iElaps = cpuSecond() - iStart;
    hipMemcpy(odata_host, odata_dev, grid.x * sizeof(int), hipMemcpyDeviceToHost);
    gpu_sum = 0;
    for (int i = 0; i < grid.x; i++)
        gpu_sum += odata_host[i];
    printf("gpu reduceNeighboredLess   elapsed %lf ms gpu_sum: %d<<<grid %d block %d>>>\n", iElaps, gpu_sum, grid.x, block.x);

    //kernel 3:reduceInterleaved
    CHECK(hipMemcpy(idata_dev, idata_host, nBytes, hipMemcpyHostToDevice));
    CHECK(hipDeviceSynchronize());
    iStart = cpuSecond();
    reduceInterleaved << <grid, block >> >(idata_dev, odata_dev, size);
    hipDeviceSynchronize();
    iElaps = cpuSecond() - iStart;
    hipMemcpy(odata_host, odata_dev, grid.x * sizeof(int), hipMemcpyDeviceToHost);
    gpu_sum = 0;
    for (int i = 0; i < grid.x; i++)
        gpu_sum += odata_host[i];
    printf("gpu reduceInterleaved      elapsed %lf ms gpu_sum: %d<<<grid %d block %d>>>\n", iElaps, gpu_sum, grid.x, block.x);



    //kernel 4:reduceUnrolling2
    CHECK(hipMemcpy(idata_dev, idata_host, nBytes, hipMemcpyHostToDevice));
    CHECK(hipDeviceSynchronize());
    iStart = cpuSecond();
    reduceUnroll2 <<<grid.x/2, block >>>(idata_dev, odata_dev, size);       // 由于合并了一半的线程块，这里的网格个数都要对应的减少一半
    hipDeviceSynchronize();
    iElaps = cpuSecond() - iStart;
    hipMemcpy(odata_host, odata_dev, grid.x * sizeof(int), hipMemcpyDeviceToHost);
    gpu_sum = 0;
    for (int i = 0; i < grid.x/2; i++)
        gpu_sum += odata_host[i];
    printf("gpu reduceUnrolling2       elapsed %lf ms gpu_sum: %d<<<grid %d block %d>>>\n", iElaps, gpu_sum, grid.x/2, block.x);


    //kernel 5:reduceUnrolling4
    CHECK(hipMemcpy(idata_dev, idata_host, nBytes, hipMemcpyHostToDevice));
    CHECK(hipDeviceSynchronize());
    iStart = cpuSecond();
    reduceUnroll4 <<<grid.x/4, block >>>(idata_dev, odata_dev, size);       // 由于每次合并4个线程块，这里的网格个数都要对应的要变为1、4
    hipDeviceSynchronize();
    iElaps = cpuSecond() - iStart;
    hipMemcpy(odata_host, odata_dev, grid.x * sizeof(int), hipMemcpyDeviceToHost);
    gpu_sum = 0;
    for (int i = 0; i < grid.x/4; i++)
        gpu_sum += odata_host[i];
    printf("gpu reduceUnrolling4       elapsed %lf ms gpu_sum: %d<<<grid %d block %d>>>\n", iElaps, gpu_sum, grid.x/4, block.x);


    //kernel 5:reduceUnrolling8
    CHECK(hipMemcpy(idata_dev, idata_host, nBytes, hipMemcpyHostToDevice));
    CHECK(hipDeviceSynchronize());
    iStart = cpuSecond();
    reduceUnroll8 <<<grid.x/8, block >>>(idata_dev, odata_dev, size);       // 由于每次合并8个线程块，这里的网格个数都要对应的要变为1/8
    hipDeviceSynchronize();
    iElaps = cpuSecond() - iStart;
    hipMemcpy(odata_host, odata_dev, grid.x * sizeof(int), hipMemcpyDeviceToHost);
    gpu_sum = 0;
    for (int i = 0; i < grid.x/8; i++)
        gpu_sum += odata_host[i];
    printf("gpu reduceUnrolling8       elapsed %lf ms gpu_sum: %d<<<grid %d block %d>>>\n", iElaps, gpu_sum, grid.x/8, block.x);


    //kernel 6:reduceUnrollingWarp8
    CHECK(hipMemcpy(idata_dev, idata_host, nBytes, hipMemcpyHostToDevice));
    CHECK(hipDeviceSynchronize());
    iStart = cpuSecond();
    reduceUnrollWarp8 <<<grid.x/8, block >>>(idata_dev, odata_dev, size);       // 由于每次合并8个线程块，这里的网格个数都要对应的要变为1/8
    hipDeviceSynchronize();
    iElaps = cpuSecond() - iStart;
    hipMemcpy(odata_host, odata_dev, grid.x * sizeof(int), hipMemcpyDeviceToHost);
    gpu_sum = 0;
    for (int i = 0; i < grid.x/8; i++)
        gpu_sum += odata_host[i];
    printf("gpu reduceUnrollingWarp8   elapsed %lf ms gpu_sum: %d<<<grid %d block %d>>>\n", iElaps, gpu_sum, grid.x/8, block.x);


    //kernel 7:reduceCompleteUnrollWarp8
    CHECK(hipMemcpy(idata_dev, idata_host, nBytes, hipMemcpyHostToDevice));
    CHECK(hipDeviceSynchronize());
    iStart = cpuSecond();
    reduceCompleteUnrollWarp8 <<<grid.x/8, block >>>(idata_dev, odata_dev, size);       // 由于每次合并8个线程块，这里的网格个数都要对应的要变为1/8
    hipDeviceSynchronize();
    iElaps = cpuSecond() - iStart;
    hipMemcpy(odata_host, odata_dev, grid.x * sizeof(int), hipMemcpyDeviceToHost);
    gpu_sum = 0;
    for (int i = 0; i < grid.x/8; i++)
        gpu_sum += odata_host[i];
    printf("gpu reducCompleteUnrollingWarp8        elapsed %lf ms gpu_sum: %d<<<grid %d block %d>>>\n", iElaps, gpu_sum, grid.x/8, block.x);


    free(idata_host);
    free(odata_host);
    CHECK(hipFree(idata_dev));
    CHECK(hipFree(odata_dev));

    //reset device
    hipDeviceReset();

    //check the results
    if (gpu_sum == cpu_sum)
    {
        printf("Test success!\n");
    }
    return EXIT_SUCCESS;

}