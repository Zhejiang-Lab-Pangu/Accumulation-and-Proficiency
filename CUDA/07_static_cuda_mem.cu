//
// Created by zjlab on 9/11/23.
//
#include <iostream>
using namespace std;
#include <mma.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
//using namespace nvcuda;


/*
 * CUDA 的静态全局内存
 *
 * 三个接口：
 *      (1) cudaMemcpyToSymbol;
 *      (2) cudaMemcpyFromSymbol;
 *      (3) cudaGetSymbolAddress;
 *
 * */



__device__ float devData;
__global__ void checkGlobalVariable(void){
    printf("Device: The value of the global variable is %f\n", devData);
    devData += 2.0;
}

void demo0(){
    float value = 3.14f;

    // 静态H2D内存搬运需要用此接口API
    hipMemcpyToSymbol(HIP_SYMBOL(devData), &value, sizeof(float));
    printf("Host: copy %f to the global variable\n",value);

    checkGlobalVariable<<<1, 1>>>();

    // 静态D2H内存搬运需要用此接口API
    hipMemcpyFromSymbol(&value, HIP_SYMBOL(devData), sizeof(float));
    printf("Host: the value changed by the kernel to %f \n",value);

    hipDeviceReset();

}


// 非用cudaMemcpy
void demo1(){
    float value = 3.14f;

    // 首先得到devData的动态地址
    float *dptr = nullptr;
    hipGetSymbolAddress((void **)&dptr, HIP_SYMBOL(devData));


    hipMemcpy(dptr, &value, sizeof(float), hipMemcpyHostToDevice);
    printf("Host: copy %f to the global variable\n",value);

    checkGlobalVariable<<<1, 1>>>();

    hipMemcpy(&value, dptr, sizeof(float), hipMemcpyDeviceToHost);
    printf("Host: the value changed by the kernel to %f \n",value);

    hipDeviceReset();
}

int main()
{
    demo0();
    cout << endl;
    demo1();
    return 0;
}