//
// Created by zjlab on 10/7/23.
//

#include <iostream>
using namespace std;
#include "hipsparse.h"
#include "util.h"
#include "hip/hip_runtime.h"
#include "hipblas.h"
#include "hipblas.h"


/*
 * cuBLAS的数据类型
 *      1. hipblasOperation_t
 *          (1) HIPBLAS_OP_N         non-transpose operation
 *          (2) HIPBLAS_OP_T         transpose operation
 *          (3) HIPBLAS_OP_C         conjugate transpose operation
 *
 *
 * */

void demo(){

    initDevice(0);

    int M=4, K=4, N=4;
    float *lhs = new float[M*K];
    float *rhs = new float[K*N];
    float *res = new float[M*N];
    initialData(lhs, M*K);
    initialData(rhs, K*N);
    memset(res, 0, M*N);

    float *h_res = new float[M*N];
    cpuGEMM(M, K, N, lhs, rhs,h_res);

    // 分配设备内存
    float *d_lhs = nullptr;
    float *d_rhs = nullptr;
    float *d_res = nullptr;
    hipMalloc((void**)&d_lhs, sizeof(float)*M*K);
    hipMalloc((void**)&d_rhs, sizeof(float)*K*N);
    hipMalloc((void**)&d_res, sizeof(float)*M*N);


    // 创建cublas句柄
    hipblasHandle_t handle;
    hipblasCreate(&handle);


    // H2D
    hipblasSetMatrix(M, K, sizeof(float), lhs, M, d_lhs, M);
    hipblasSetMatrix(K, N, sizeof(float), rhs, K, d_rhs, K);


    // Sgemm
    const float alpha = 1.0;
    const float beta = 0.0;
    hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, M, N, K, &alpha, d_lhs, K, d_rhs, N, &beta, d_res, M);        //返回的是列主序的结果矩阵（转置矩阵）

    // D2H
    hipblasGetMatrix(M, N, sizeof(float), d_res, M, res, M);


    printMatrix(res, M, N);

    cout << endl;

    printMatrix(h_res, M, N);


    hipFree(d_lhs);
    hipFree(d_res);
    hipFree(d_rhs);
    delete [] lhs;
    delete [] rhs;
    delete [] res;
    delete [] h_res;

}

int main(){
    demo();
    return 0;
}